// ================================================================================================
// Tim Backus
// CIS 450 - High Performance Computing
// 3D Game of Life - CUDA Version
// ================================================================================================

#define GOL_IO_FILENAME "gol3DOutput.dat"
#define GOL_CUDA_THREADS_SIZE 8


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <time.h>
#include <stdlib.h>

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Gather & Map) - Adds up the number of neighbors for a cell in a 3x3x3 cube and
//   sets each cell to alive or dead depending on its number of neighbors and the rules for this 
//   current game.
// ------------------------------------------------------------------------------------------------
__global__
void lifeItrKernel(const char* const d_in, char* d_out, const unsigned int xsize,
                   const unsigned int ysize, const unsigned int zsize, const unsigned int alow,
                   const unsigned int ahigh) {
  
  extern __shared__ char shMem[];
               
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const unsigned int stepX = ysize * zsize;
  const unsigned int arrayPos = threadPosX * stepX + threadPosY * zsize + threadPosZ;
  const unsigned int threadID = threadIdx.x * stepX + threadIdx.y * zsize + threadIdx.z;

  // Ensure thread bounds
  if(threadPosX > xsize - 1) return;
  if(threadPosY > ysize - 1) return;
  if(threadPosZ > zsize - 1) return;
  
  // Copy global into shared memory
  shMem[threadID] = d_in[arrayPos];
  __syncthreads();
  
  // Begin adding neighbors
  char sum = 0;
  
  // X-Axis neighbors
  int xc, xcoord;
  for(xc = threadPosX - 1; xc <= threadPosX + 1; xc++) {
    
    // Wrap X-Axis
    xcoord = xc;
    if(xc < 0) xcoord = xsize;
    else if(xc >= xsize) xcoord = 0;
    
    // Y-Axis neighbors
    int yc, ycoord;
    for(yc = threadPosY - 1; yc <= threadPosY + 1; yc++) {
      
      // Wrap Y-Axis
      ycoord = yc;
      if(yc < 0) ycoord = ysize;
      else if(yc >= ysize) ycoord = 0;
      
      // Z-Axis neighbors
      int zc, zcoord;
      for(zc = threadPosZ - 1; zc <= threadPosZ + 1; zc++) {
        
        // Wrap Z-Axis
        zcoord = zc;
        if(zc < 0) zcoord = zsize;
        else if(zc >= zsize) zcoord = 0;
        
        // Don't count the cell itself
        if(threadPosX != xcoord || threadPosY != ycoord || threadPosZ != zcoord) {
          
          // Use shared memory instead of global memory if the current coord is in the thread block
          //   (is all of this overhead even worth it?)
          if((xcoord >= blockDim.x * blockIdx.x && xcoord < (blockDim.x + 1) * blockIdx.x) &&
             (ycoord >= blockDim.y * blockIdx.y && ycoord < (blockDim.y + 1) * blockIdx.y) &&
             (zcoord >= blockDim.z * blockIdx.z && zcoord < (blockDim.z + 1) * blockIdx.z)) {
            sum += shMem[threadID];
          } else {
            sum += d_in[xcoord * stepX + ycoord * zsize + zcoord];
          }
        }
      }
    }
  }
  
  // Set the cell's dead or alive status based on its neighbor count
  if (sum < alow || sum > ahigh) {
    d_out[arrayPos] = 0;
  } else if (sum >= alow && sum <= ahigh) {
    d_out[arrayPos] = 1;
  }
  
}


// ------------------------------------------------------------------------------------------------
// CUDA kernel (Gather) - Adds up the number of neighbors for a cell in a 3x3x3 cube.
// ------------------------------------------------------------------------------------------------
__global__
void sumNeighborsKernel(const char* const d_in, char* d_out, const unsigned int xsize,
                        const unsigned int ysize, const unsigned int zsize) {
  
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const unsigned int stepX = ysize * zsize;
  const unsigned int arrayPos = threadPosX * stepX + threadPosY * zsize + threadPosZ;
  
  // printf("TID=%d,%d,%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
  // printf("TPOS=%d,%d,%d\n", threadPosX, threadPosY, threadPosZ);
  // printf("APOS=%d\n", arrayPos);
  
  // Ensure thread bounds
  if(threadPosX > xsize - 1) return;
  if(threadPosY > ysize - 1) return;
  if(threadPosZ > zsize - 1) return;
  
  char sum = 0;
  
  // X-Axis neighbors
  int xc, xcoord;
  for(xc = threadPosX - 1; xc <= threadPosX + 1; xc++) {
    
    // Wrap X-Axis
    xcoord = xc;
    if(xc < 0) xcoord = xsize;
    else if(xc >= xsize) xcoord = 0;
    
    // Y-Axis neighbors
    int yc, ycoord;
    for(yc = threadPosY - 1; yc <= threadPosY + 1; yc++) {
      
      // Wrap Y-Axis
      ycoord = yc;
      if(yc < 0) ycoord = ysize;
      else if(yc >= ysize) ycoord = 0;
      
      // Z-Axis neighbors
      int zc, zcoord;
      for(zc = threadPosZ - 1; zc <= threadPosZ + 1; zc++) {
        
        // Wrap Z-Axis
        zcoord = zc;
        if(zc < 0) zcoord = zsize;
        else if(zc >= zsize) zcoord = 0;
        
        // Don't count the cell itself
        if(threadPosX != xcoord || threadPosY != ycoord || threadPosZ != zcoord) {
          sum += d_in[xcoord * stepX + ycoord * zsize + zcoord];
        }
        
      }
    }
  }
  
  d_out[arrayPos] = sum;
}

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Map) - Sets each cell to alive or dead depending on its number of neighbors and
//   the rules for this current game.
// ------------------------------------------------------------------------------------------------
__global__
void setAliveDeadKernel(const char* const d_nei, char* d_out, const unsigned int xs, 
                        const unsigned int ys, const unsigned int zs, const unsigned int alow, 
                        const unsigned int ahigh) {
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const int stepX = ys * zs;
  const int arrayPos = threadPosX * stepX + threadPosY * zs + threadPosZ;

  // Ensure thread bounds
  if(threadPosX > xs - 1) return;
  if(threadPosY > ys - 1) return;
  if(threadPosZ > zs - 1) return;
  
  // Set the cell alive or dead as according to the rules
  if (d_nei[arrayPos] < alow || d_nei[arrayPos] > ahigh) {
    d_out[arrayPos] = 0;
  } else if (d_nei[arrayPos] >= alow && d_nei[arrayPos] <= ahigh) {
    d_out[arrayPos] = 1;
  }
}

// ------------------------------------------------------------------------------------------------
// Returns the 1D position of a simulated 3D array
// ------------------------------------------------------------------------------------------------
int getArrIndex(const unsigned int xp, const unsigned int yp, const unsigned int zp,
                const unsigned int ys, const unsigned int zs) {
  return xp * ys * zs + yp * zs + zp;
}

// ------------------------------------------------------------------------------------------------
// Prints a 3D array.
// ------------------------------------------------------------------------------------------------
void print3DArray(char* arr, unsigned const int x, unsigned const int y, unsigned const int z) {
  int i;
  for(i = 0; i < x; ++i) {
    printf("Dimension %d:\n", i);
    int j;
    for(j = 0; j < y; ++j) {
      int k;
      for(k = 0; k < z; ++k) {
        printf("%d ", (char)arr[getArrIndex(i, j, k, y, z)]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

// ------------------------------------------------------------------------------------------------
// Writes cells to alive or dead, randomly.
// ------------------------------------------------------------------------------------------------
void randomizeGrid(char* grid, unsigned const int size, unsigned const int chance) {
 
  srand(time(NULL));
  int i;
  for(i = 0; i < size; i++) {
    grid[i] = (char)((rand() % 100 <= chance) ? 1 : 0);
  }
  
}

// ------------------------------------------------------------------------------------------------
// Initializes the game data file.
// Line 1: <iteration count> <x-size> <y-size> <z-size>
// Line 2: Blank
// ------------------------------------------------------------------------------------------------
void initGameFile(const unsigned int itrs, const unsigned int x, const unsigned int y,
                  const unsigned int z) {
  FILE *fp;
  fp = fopen(GOL_IO_FILENAME, "w+");
  fprintf(fp, "%d %d %d %d\n\n", itrs, x, y, z);
  fclose(fp);
}

// ------------------------------------------------------------------------------------------------
// Writes a game to a file for visualization within Java.
// For every iteration, a block of text is created of the format:
//   "<x-coord>:<z-coords for y=0>, <z-coords for y=1>, ..."
//   Z-coords are represented by a 0 or 1 for each z-coordinate
// Example: Game with 5 iterations, x=3, y=7, z=4
//   5 3 7 4
//   
//   0:0000,0000,0000,0000,0000,0000,0000
//   1:0000,0000,0000,0100,0000,0000,0001
//   2:0000,0000,0010,0100,0001,0011,0000
//   0:0000,0000,0000,0000,0000,0000,0000
//   1:0000,0000,0000,0100,0000,0000,0001
//   2:0000,0000,0010,0100,0001,0011,0000
//   0:0000,0000,0000,0000,0000,0000,0000
//   1:0000,0000,0000,0100,0000,0000,0001
//   2:0000,0000,0010,0100,0001,0011,0000
//   0:0000,0000,0000,0000,0000,0000,0000
//   1:0000,0000,0000,0100,0000,0000,0001
//   2:0000,0000,0010,0100,0001,0011,0000
//   0:0000,0000,0000,0000,0000,0000,0000
//   1:0000,0000,0000,0100,0000,0000,0001
//   2:0000,0000,0010,0100,0001,0011,0000
// 
// ------------------------------------------------------------------------------------------------
void writeGameStep(char* arr, unsigned const int x, unsigned const int y, unsigned const int z) {
  FILE *fp;
  fp = fopen(GOL_IO_FILENAME, "a");
  
  int i;
  for(i = 0; i < x; i++) {
    fprintf(fp, "%d:", i);
    int j;
    for(j = 0; j < y; j++) {
      
      if(j > 0) {
        fprintf(fp, ",");
      }
      // Print Z-Dim values
      int k;
      for(k = 0; k < z; k++) {
        fprintf(fp, "%d", arr[getArrIndex(i, j, k, y, z)]);
      }
    }
    fprintf(fp, "\n");
  }
  fclose(fp);
}

// ------------------------------------------------------------------------------------------------
// Runs the Game of Life.
// ------------------------------------------------------------------------------------------------
void runLife(const unsigned int iterations, unsigned int xsize, const unsigned int ysize, 
             const unsigned int zsize, const unsigned int initc, const unsigned int alow,
             const unsigned int ahigh, const unsigned int printArr, const unsigned int writeOut) {
  
  // Memory values
  const unsigned int arrSize = xsize * ysize * zsize;
  const unsigned int arrMem = arrSize * sizeof(char);
  
  // GPU grid dimensions
  const int gx = ceil((double) xsize / GOL_CUDA_THREADS_SIZE);
  const int gy = ceil((double) ysize / GOL_CUDA_THREADS_SIZE);
  const int gz = ceil((double) zsize / GOL_CUDA_THREADS_SIZE);
  printf("Grid dimension: %d,%d,%d\n", gx, gy, gz);
  dim3 gridDim(gx, gy, gz);
  
  // GPU thread dimensions
  const int tx = GOL_CUDA_THREADS_SIZE;
  const int ty = GOL_CUDA_THREADS_SIZE;
  const int tz = GOL_CUDA_THREADS_SIZE;
  printf("Block dimension: %d,%d,%d\n", tx, ty, tz);
  dim3 blockDim(tx, ty, tz);
  
  // Initialize game space
  char *h_in = (char *) malloc(arrMem);
  printf("Randomizing initial game (could take a while)...\n");
  randomizeGrid(h_in, arrSize, initc);
  
  // Print the initial array if enabled
  if(printArr) {
    printf("Initial grid:\n");
    print3DArray(h_in, xsize, ysize, zsize);
  }
  
  // Initialize the output file if enabled
  if(writeOut) {
    initGameFile(iterations, xsize, ysize, zsize);
  }
  
  // Number of neighbors
  char *h_nei = (char *) malloc(arrMem);
  
  // Pointers for GPU game data
  char *d_in;
  char *d_out;
  
  // Allocate input array on GPU
  printf("Allocating %d bytes of memory on the GPU...\n", 
         (int)(xsize * ysize * zsize * sizeof(char)));
  hipMalloc(&d_in, arrMem);
  
  // Allocate output array on GPU
  hipMalloc(&d_out, arrMem);
  
  // Copy the host data to the GPU
  hipMemcpy(d_in, h_in, arrMem, hipMemcpyHostToDevice);
  
  // Do Game of Life iterations
  int itrNum;
  for(itrNum = 0; itrNum < iterations; itrNum++) {
    
    printf("Iteration %d ", itrNum);
    
    // Run the kernel to simulate an iteration of 3D life
    clock_t start = clock();
    lifeItrKernel<<<gridDim, blockDim, arrMem>>>(d_in, d_out, xsize, ysize, zsize, alow, ahigh);
    hipError_t cerr = hipDeviceSynchronize();
    if(cerr != hipSuccess) {
      printf("Kernel sumNeighbors failed with error \"%s\".\n", hipGetErrorString(cerr));
    }
    clock_t end = clock();
    
    // Copy the memory back to the input array for the next iteration
    hipMemcpy(d_in, d_out, arrMem, hipMemcpyDeviceToDevice);
    
    printf("took %d ticks.\n", (end - start));
    if(printArr || writeOut) {
      hipMemcpy(h_in, d_out, arrMem, hipMemcpyDeviceToHost);
      if(printArr) {
        print3DArray(h_in, xsize, ysize, zsize);
      }
      if(writeOut) {
        printf("  Writing iteration to file...\n");
        writeGameStep(h_in, xsize, ysize, zsize);
      }
    }
  }
  
  // Free memory
  hipFree(d_in);
  hipFree(d_out);
  free(h_in);
}

// ------------------------------------------------------------------------------------------------
// Prints the usage message if a bad number of runtime arguments are passed.
// ------------------------------------------------------------------------------------------------
void printUsage() {
  printf("Arguments (separated by spaces):\n");
  printf("  MAX_ITERATIONS\n  SIZE_X\n  SIZE_Y\n  SIZE_Z\n  INITIAL_ALIVE_CHANCE (1-100)\n");
  printf("  ALIVE_THRESHOLD_LOW (inclusive)\n  ALIVE_THRESHOLD_HIGH (inclusive)\n");
  printf("  PRINT_ARRAY? (0=no, 1=yes)\n  WRITE_TO_FILE? (0=no, 1=yes)\n");
}

// ------------------------------------------------------------------------------------------------
// Main Method
// ------------------------------------------------------------------------------------------------
int main(int argc, char *argv[]) {
  
  // Ensure proper runtime argument count
  if(argc != 10) {
    printUsage();
    return EXIT_SUCCESS;
  }
  
  // Parse iteration count
  unsigned const int iterations = atoi(argv[1]);
  
  // Parse X-Size
  unsigned const int sizeX = atoi(argv[2]);
  
  // Parse Y-Size
  unsigned const int sizeY = atoi(argv[3]);
  
  // Parse Z-Size
  unsigned const int sizeZ = atoi(argv[4]);
  
  // Parse initial alive chance
  unsigned const int initChance = atoi(argv[5]);
  
  // Parse alive low threshold (inclusive)
  unsigned const int aliveLow = atoi(argv[6]);
  
  // Parse alive high threshold (inclusive)
  unsigned const int aliveHigh = atoi(argv[7]);
  
  // Parse whether or not to print the array
  unsigned const int printArray = atoi(argv[8]);
  
  // Parse whether or not to output to disk
  unsigned const int writeOut = atoi(argv[9]);
  
  // Print game information to the console
  printf("Starting %d iteration Game of Life (CUDA) with sizes x=%d, y=%d, z=%d\n", iterations,
         sizeX, sizeY, sizeZ);
  printf("  initial alive chance=%d, neighbors for alive=%d to %d\n", initChance, 
         aliveLow, aliveHigh);
  if(writeOut) {
    printf("  File output enabled.\n");
  }
  runLife(iterations, sizeX, sizeY, sizeZ, initChance, aliveLow, aliveHigh, printArray, writeOut);
  
  return EXIT_SUCCESS;
}