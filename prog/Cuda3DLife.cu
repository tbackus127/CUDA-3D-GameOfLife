// ================================================================================================
// Tim Backus
// CIS 450 - High Performance Computing
// 3D Game of Life - CUDA Version
// ================================================================================================

#define GOL_IO_FILENAME "gol3DOutput.dat"
#define GOL_CUDA_THREADS_PER_BLOCK 32


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <time.h>
#include <stdlib.h>

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Gather) - Adds up the number of neighbors for a cell in a 3x3x3 cube.
// ------------------------------------------------------------------------------------------------
__global__
void sumNeighborsKernel(const char* const d_in, char* d_out, const unsigned int xs,
                        const unsigned int ys, const unsigned int zs) {
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const int stepX = xs * ys;
  const int arrayPos = threadPosX * stepX + threadPosY * ys + threadPosZ;
  
  // Ensure thread bounds
  if(threadPosX > xs - 1) return;
  if(threadPosY > ys - 1) return;
  if(threadPosZ > zs - 1) return;
  
  // X-Axis neighbors
  unsigned char sum = 0;
  int i, realx;
  for(i = arrayPos - stepX; i <= arrayPos + stepX; i += stepX) {
    
    // Wrap X-Axis
    realx = i;
    if(i > xs) {
      realx = threadPosY * ys + threadPosZ;
    } else if(i < 0) {
      realx = (xs - 1) * stepX + threadPosY * ys + threadPosZ;
    }
    
    // Y-Axis neighbors
    int j, realy;
    for(j = arrayPos - ys; j <= arrayPos + ys; j += ys) {
      
      // Wrap Y-Axis
      realy = j;
      if(j > ys) {
        realy = threadPosZ;
      } else if(j < 0) {
        realy = (ys - 1) * ys + threadPosZ;
      }
      
      // Z-Axis neighbors
      int k, realz;
      for(k = arrayPos - 1; k <= arrayPos + 1; k++) {
        
        // Wrap Z-Axis
        realz = k;
        if(k > zs) {
          realz = 0;
        } else if(k < 0) {
          realz = zs - 1;
        }
        
        sum += d_in[realx * stepX + realy * ys + realz];
      }
    }
  }
  
  d_out[arrayPos] = sum;
}

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Map) - Sets each cell to alive or dead depending on its number of neighbors and
//   the rules for this current game.
// ------------------------------------------------------------------------------------------------
__global__
void setAliveDeadKernel(const char* const d_nei, char* d_out, const unsigned int xs, 
                        const unsigned int ys, const unsigned int zs, const unsigned int alow, 
                        const unsigned int ahigh) {
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const int stepX = xs * ys;
  const int arrayPos = threadPosX * stepX + threadPosY * ys + threadPosZ;
  
  // Ensure thread bounds
  if(threadPosX > xs - 1) return;
  if(threadPosY > ys - 1) return;
  if(threadPosZ > zs - 1) return;
  
  // Set the cell alive or dead as according to the rules
  if (d_nei[arrayPos] < alow || d_nei[arrayPos] > ahigh) {
    d_out[arrayPos] = 0;
  } else if (d_nei[arrayPos] >= alow && d_nei[arrayPos] <= ahigh) {
    d_out[arrayPos] = 1;
  }
}

// ------------------------------------------------------------------------------------------------
// Returns the 1D position of a simulated 3D array
// ------------------------------------------------------------------------------------------------
int getArrIndex(const unsigned int xp, const unsigned int yp, const unsigned int zp,
                const unsigned int xs, const unsigned int ys) {
  return xp * xs * ys + ys * yp + zp;
}

// ------------------------------------------------------------------------------------------------
// Writes cells to alive or dead, randomly.
// ------------------------------------------------------------------------------------------------
void randomizeGrid(char* grid, unsigned const int size, unsigned const int chance) {
 
  srand(time(NULL));
  int i;
  for(i = 0; i < size; i++) {
    grid[i] = (char)((rand() % 100 <= chance) ? 1 : 0);
  }
  
}

// ------------------------------------------------------------------------------------------------
// Runs the Game of Life.
// ------------------------------------------------------------------------------------------------
void runLife(const unsigned int iterations, const unsigned int xsize, const unsigned int ysize, 
             const unsigned int zsize, const unsigned int initc, const unsigned int alow,
             const unsigned int ahigh, const unsigned int outputToFile) {
  
  // Memory values
  const int arrSize = xsize * ysize * zsize;
  const int arrMem = arrSize * sizeof(char);
  
  // GPU grid dimensions
  const int gx = (int) ceil(xsize / GOL_CUDA_THREADS_PER_BLOCK);
  const int gy = (int) ceil(ysize / GOL_CUDA_THREADS_PER_BLOCK);
  const int gz = (int) ceil(zsize / GOL_CUDA_THREADS_PER_BLOCK);
  dim3 gridDim(gx, gy, gz);
  
  // GPU thread dimensions
  const int tx = (xsize >= GOL_CUDA_THREADS_PER_BLOCK) ? GOL_CUDA_THREADS_PER_BLOCK : xsize;
  const int ty = (ysize >= GOL_CUDA_THREADS_PER_BLOCK) ? GOL_CUDA_THREADS_PER_BLOCK : ysize;
  const int tz = (zsize >= GOL_CUDA_THREADS_PER_BLOCK) ? GOL_CUDA_THREADS_PER_BLOCK : zsize;
  dim3 blockDim(tx, ty, tz);
  
  // Initialize game space
  char *h_in = (char *) malloc(arrMem);
  randomizeGrid(h_in, arrSize, initc);
  
  // Number of neighbors
  char *h_nei = (char *) malloc(arrMem);

  // Allocate X-Size on GPU
  int d_xs;
  hipMalloc((void **) &d_xs, sizeof(int));
  hipMemcpy(&d_xs, &xsize, sizeof(int), hipMemcpyHostToDevice);
  
  // Allocate Y-Size on GPU
  int d_ys;
  hipMalloc((void **) &d_ys, sizeof(int));
  hipMemcpy(&d_ys, &ysize, sizeof(int), hipMemcpyHostToDevice);
  
  // Allocate Z-Size on GPU
  int d_zs;
  hipMalloc((void **) &d_zs, sizeof(int));
  hipMemcpy(&d_zs, &zsize, sizeof(int), hipMemcpyHostToDevice);
  
  // Allocate neighbor count for alive low threshold on GPU
  int d_lw;
  hipMalloc((void **) &d_lw, sizeof(int));
  hipMemcpy(&d_lw, &alow, sizeof(int), hipMemcpyHostToDevice);
  
  // Allocate neighbor count for alive low threshold on GPU
  int d_hg;
  hipMalloc((void **) &d_hg, sizeof(int));
  hipMemcpy(&d_hg, &ahigh, sizeof(int), hipMemcpyHostToDevice);
  
  // Pointers for GPU game data
  char *d_in;
  char *d_out;
  
  // Allocate input array on GPU
  hipMalloc(&d_in, arrMem);
  
  // Allocate output array on GPU
  hipMalloc(&d_out, arrMem);
  
  // Do Game of Life iterations
  int itrNum;
  for(itrNum = 0; itrNum < iterations; itrNum++) {
    
    printf("Iteration %d ", itrNum);
    
    clock_t start = clock();
    
    // Run the kernel to add neighbors of all cells
    hipMemcpy(d_in, h_in, arrMem, hipMemcpyHostToDevice);
    sumNeighborsKernel<<<gridDim, blockDim>>>(d_in, d_out, d_xs, d_ys, d_zs);
    hipMemcpy(h_nei, d_out, arrMem, hipMemcpyDeviceToHost);
    
    // Run the kernel to set the cells' alive or dead states
    hipMemcpy(d_in, h_nei, arrMem, hipMemcpyHostToDevice);
    setAliveDeadKernel<<<gridDim, blockDim>>>(d_in, d_out, d_xs, d_ys, d_zs, d_lw, d_hg);
    hipMemcpy(h_in, d_out, arrMem, hipMemcpyDeviceToHost);
    
    clock_t end = clock();
    
    printf("took %d ticks.\n", (end - start));
  }
  
  // Free memory
  hipFree(d_in);
  hipFree(d_out);
  hipFree(&d_xs);
  hipFree(&d_ys);
  hipFree(&d_zs);
  hipFree(&d_lw);
  hipFree(&d_hg);
  free(h_in);
}

// ------------------------------------------------------------------------------------------------
// Prints the usage message if a bad number of runtime arguments are passed.
// ------------------------------------------------------------------------------------------------
void printUsage() {
  printf("Usage: <program> MAX_ITERATIONS, SIZE_X, SIZE_Y, SIZE_Z,\nINITIAL_ALIVE_CHANCE, ");
  printf("  ALIVE_THRESHOLD_LOW (inclusive), ALIVE_THRESHOLD_HIGH (inclusive)");
}

// ------------------------------------------------------------------------------------------------
// Main Method
// ------------------------------------------------------------------------------------------------
int main(int argc, char *argv[]) {
  
  // Ensure proper runtime argument count
  if(argc <= 1 || argc > 9) {
    printUsage();
    return EXIT_SUCCESS;
  }
  
  // Parse iteration count
  unsigned const int iterations = atoi(argv[1]);
  
  // Parse X-Size
  unsigned const int sizeX = atoi(argv[2]);
  
  // Parse Y-Size
  unsigned const int sizeY = atoi(argv[3]);
  
  // Parse Z-Size
  unsigned const int sizeZ = atoi(argv[4]);
  
  // Parse initial alive chance
  unsigned const int initChance = atoi(argv[5]);
  
  // Parse alive low threshold (inclusive)
  unsigned const int aliveLow = atoi(argv[6]);
  
  // Parse alive high threshold (inclusive)
  unsigned const int aliveHigh = atoi(argv[7]);
  
  // Parse whether or not to output to file (0 or 1)
  unsigned const int outputEnabled = atoi(argv[8]);
  
  printf("Starting %d iteration Game of Life (CUDA) with sizes x=%d, y=%d, z=%d\n", iterations,
         sizeX, sizeY, sizeZ);
  runLife(iterations, sizeX, sizeY, sizeZ, initChance, aliveLow, aliveHigh, outputEnabled);
  
  return EXIT_SUCCESS;
}