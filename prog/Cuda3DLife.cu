// ================================================================================================
// Tim Backus
// CIS 450 - High Performance Computing
// 3D Game of Life - CUDA Version
// ================================================================================================

#define GOL_IO_FILENAME "gol3DOutput.dat"
#define GOL_CUDA_THREADS_PER_BLOCK 8


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <time.h>
#include <stdlib.h>

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Gather) - Adds up the number of neighbors for a cell in a 3x3x3 cube.
// ------------------------------------------------------------------------------------------------
__global__
void sumNeighborsKernel(const char* const d_in, char* d_out, const unsigned int xsize,
                        const unsigned int ysize, const unsigned int zsize) {
  
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const unsigned int stepX = ysize * zsize;
  const unsigned int arrayPos = threadPosX * stepX + threadPosY * zsize + threadPosZ;
  
  // printf("TID=%d,%d,%d\n", threadIdx.x, threadIdx.y, threadIdx.z);
  // printf("TPOS=%d,%d,%d\n", threadPosX, threadPosY, threadPosZ);
  // printf("APOS=%d\n", arrayPos);
  
  // Ensure thread bounds
  if(threadPosX > xsize - 1) return;
  if(threadPosY > ysize - 1) return;
  if(threadPosZ > zsize - 1) return;
  
  char sum = 0;
  
  // X-Axis neighbors
  int xc, xcoord;
  for(xc = threadPosX - 1; xc <= threadPosX + 1; xc++) {
    
    // Wrap X-Axis
    xcoord = xc;
    if(xc < 0) xcoord = xsize;
    else if(xc >= xsize) xcoord = 0;
    
    // Y-Axis neighbors
    int yc, ycoord;
    for(yc = threadPosY - 1; yc <= threadPosY + 1; yc++) {
      
      // Wrap Y-Axis
      ycoord = yc;
      if(yc < 0) ycoord = ysize;
      else if(yc >= ysize) ycoord = 0;
      
      // Z-Axis neighbors
      int zc, zcoord;
      for(zc = threadPosZ - 1; zc <= threadPosZ + 1; zc++) {
        
        // Wrap Z-Axis
        zcoord = zc;
        if(zc < 0) zcoord = zsize;
        else if(zc >= zsize) zcoord = 0;
        
        // Don't count the cell itself
        if(threadPosX != xcoord || threadPosY != ycoord || threadPosZ != zcoord) {
          sum += d_in[xcoord * stepX + ycoord * zsize + zcoord];
        }
        
      }
    }
  }
  
  d_out[arrayPos] = sum;
}

// ------------------------------------------------------------------------------------------------
// CUDA kernel (Map) - Sets each cell to alive or dead depending on its number of neighbors and
//   the rules for this current game.
// ------------------------------------------------------------------------------------------------
__global__
void setAliveDeadKernel(const char* const d_nei, char* d_out, const unsigned int xs, 
                        const unsigned int ys, const unsigned int zs, const unsigned int alow, 
                        const unsigned int ahigh) {
  
  // Calculate block and thread IDs
  const int threadPosX = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadPosY = blockIdx.y * blockDim.y + threadIdx.y;
  const int threadPosZ = blockIdx.z * blockDim.z + threadIdx.z;
  const int stepX = ys * zs;
  const int arrayPos = threadPosX * stepX + threadPosY * zs + threadPosZ;

  // Ensure thread bounds
  if(threadPosX > xs - 1) return;
  if(threadPosY > ys - 1) return;
  if(threadPosZ > zs - 1) return;
  
  // Set the cell alive or dead as according to the rules
  if (d_nei[arrayPos] < alow || d_nei[arrayPos] > ahigh) {
    d_out[arrayPos] = 0;
  } else if (d_nei[arrayPos] >= alow && d_nei[arrayPos] <= ahigh) {
    d_out[arrayPos] = 1;
  }
}

// ------------------------------------------------------------------------------------------------
// Returns the 1D position of a simulated 3D array
// ------------------------------------------------------------------------------------------------
int getArrIndex(const unsigned int xp, const unsigned int yp, const unsigned int zp,
                const unsigned int ys, const unsigned int zs) {
  return xp * ys * zs + yp * zs + zp;
}

// ------------------------------------------------------------------------------------------------
// Prints a 3D array.
// ------------------------------------------------------------------------------------------------
void print3DArray(char* arr, unsigned const int x, unsigned const int y, unsigned const int z) {
  int i;
  for(i = 0; i < x; ++i) {
    printf("Dimension %d:\n", i);
    int j;
    for(j = 0; j < y; ++j) {
      int k;
      for(k = 0; k < z; ++k) {
        printf("%d ", (char)arr[getArrIndex(i, j, k, y, z)]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

// ------------------------------------------------------------------------------------------------
// Writes cells to alive or dead, randomly.
// ------------------------------------------------------------------------------------------------
void randomizeGrid(char* grid, unsigned const int size, unsigned const int chance) {
 
  srand(time(NULL));
  int i;
  for(i = 0; i < size; i++) {
    grid[i] = (char)((rand() % 100 <= chance) ? 1 : 0);
  }
  
}

// ------------------------------------------------------------------------------------------------
// Runs the Game of Life.
// ------------------------------------------------------------------------------------------------
void runLife(const unsigned int iterations, unsigned int xsize, const unsigned int ysize, 
             const unsigned int zsize, const unsigned int initc, const unsigned int alow,
             const unsigned int ahigh) {
  
  // Memory values
  const unsigned int arrSize = xsize * ysize * zsize;
  const unsigned int arrMem = arrSize * sizeof(char);
  
  // GPU grid dimensions
  const int gx = ceil((double) xsize / GOL_CUDA_THREADS_PER_BLOCK);
  const int gy = ceil((double) ysize / GOL_CUDA_THREADS_PER_BLOCK);
  const int gz = ceil((double) zsize / GOL_CUDA_THREADS_PER_BLOCK);
  printf("Grid dimension: %d,%d,%d\n", gx, gy, gz);
  dim3 gridDim(gx, gy, gz);
  
  // GPU thread dimensions
  const int tx = GOL_CUDA_THREADS_PER_BLOCK;
  const int ty = GOL_CUDA_THREADS_PER_BLOCK;
  const int tz = GOL_CUDA_THREADS_PER_BLOCK;
  printf("Block dimension: %d,%d,%d\n", tx, ty, tz);
  dim3 blockDim(tx, ty, tz);
  
  // Initialize game space
  char *h_in = (char *) malloc(arrMem);
  printf("Randomizing initial game (could take a while)...\n");
  randomizeGrid(h_in, arrSize, initc);
  // printf("Initial grid:\n");
  // print3DArray(h_in, xsize, ysize, zsize);
  
  // Number of neighbors
  char *h_nei = (char *) malloc(arrMem);
  
  // Pointers for GPU game data
  char *d_in;
  char *d_out;
  
  // Allocate input array on GPU
  printf("Allocating %d bytes of memory on the GPU...\n", (int)(xsize * ysize * zsize * sizeof(char)));
  hipMalloc(&d_in, arrMem);
  
  // Allocate output array on GPU
  hipMalloc(&d_out, arrMem);
  
  // Do Game of Life iterations
  int itrNum;
  for(itrNum = 0; itrNum < iterations; itrNum++) {
    
    printf("Iteration %d ", itrNum);
    
    clock_t start = clock();
    
    // Run the kernel to add neighbors of all cells
    hipMemcpy(d_in, h_in, arrMem, hipMemcpyHostToDevice);
    sumNeighborsKernel<<<gridDim, blockDim>>>(d_in, d_out, xsize, ysize, zsize);
    hipError_t cerr = hipDeviceSynchronize();
    if(cerr != hipSuccess) {
      printf("Kernel sumNeighbors failed with error \"%s\".\n", hipGetErrorString(cerr));
    }
    
    // Copy the output back to the input
    hipMemcpy(d_in, d_out, arrMem, hipMemcpyDeviceToDevice);
    
    // Run the kernel to set cells alive or dead
    setAliveDeadKernel<<<gridDim, blockDim>>>(d_in, d_out, xsize, ysize, zsize, alow, ahigh);
    cerr = hipDeviceSynchronize();
    if(cerr != hipSuccess) {
      printf("Kernel setAliveDead failed with error \"%s\".\n", hipGetErrorString(cerr));
    }
    hipMemcpy(h_in, d_out, arrMem, hipMemcpyDeviceToHost);
    
    clock_t end = clock();
    
    printf("took %d ticks.\n", (end - start));
    
    // print3DArray(h_in, xsize, ysize, zsize);
  }
  
  // Free memory
  hipFree(d_in);
  hipFree(d_out);
  free(h_in);
}

// ------------------------------------------------------------------------------------------------
// Prints the usage message if a bad number of runtime arguments are passed.
// ------------------------------------------------------------------------------------------------
void printUsage() {
  printf("Usage: <program> MAX_ITERATIONS, SIZE_X, SIZE_Y, SIZE_Z,\nINITIAL_ALIVE_CHANCE, ");
  printf("  ALIVE_THRESHOLD_LOW (inclusive), ALIVE_THRESHOLD_HIGH (inclusive)");
}

// ------------------------------------------------------------------------------------------------
// Main Method
// ------------------------------------------------------------------------------------------------
int main(int argc, char *argv[]) {
  
  // Ensure proper runtime argument count
  if(argc <= 1 || argc > 8) {
    printUsage();
    return EXIT_SUCCESS;
  }
  
  // Parse iteration count
  unsigned const int iterations = atoi(argv[1]);
  
  // Parse X-Size
  unsigned const int sizeX = atoi(argv[2]);
  
  // Parse Y-Size
  unsigned const int sizeY = atoi(argv[3]);
  
  // Parse Z-Size
  unsigned const int sizeZ = atoi(argv[4]);
  
  // Parse initial alive chance
  unsigned const int initChance = atoi(argv[5]);
  
  // Parse alive low threshold (inclusive)
  unsigned const int aliveLow = atoi(argv[6]);
  
  // Parse alive high threshold (inclusive)
  unsigned const int aliveHigh = atoi(argv[7]);
  
  printf("Starting %d iteration Game of Life (CUDA) with sizes x=%d, y=%d, z=%d\n", iterations,
         sizeX, sizeY, sizeZ);
  runLife(iterations, sizeX, sizeY, sizeZ, initChance, aliveLow, aliveHigh);
  
  return EXIT_SUCCESS;
}